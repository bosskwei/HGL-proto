#include "hip/hip_runtime.h"
#include <torch/extension.h>

template <typename index_t, typename value_t>
__global__ void propogate_move_cuda_kernel(
    index_t n_nodes, index_t n_features,
    const index_t *indptr, const index_t *indices, const value_t *features,
    value_t *out_features) {
    for (index_t row = blockIdx.x; row < n_nodes; row += gridDim.x) {
        for (index_t k = threadIdx.x; k < n_features; k += blockDim.x) {
            value_t out = 0.0;
            for (index_t i = indptr[row]; i < indptr[row + 1]; i += 1) {
                index_t col = indices[i];
                out += features[col * n_features + k];
            }
            out_features[row * n_features + k] = out;
        }
    }
}

torch::Tensor propogate_move_cuda(const torch::Tensor &indptr,
                                  const torch::Tensor &indices,
                                  const torch::Tensor &features) {
    TORCH_CHECK(indptr.dim() == 1);
    TORCH_CHECK(features.dim() == 2);
    int32_t n_nodes = indptr.size(0) - 1;
    int32_t n_features = features.size(1);

    auto new_features = torch::zeros({n_nodes, n_features}, features.options());
    propogate_move_cuda_kernel<int32_t, float><<<n_nodes, 32>>>(
        n_nodes, n_features, indptr.data_ptr<int32_t>(), indices.data_ptr<int32_t>(),
        features.data_ptr<float>(), new_features.data_ptr<float>());
    return new_features;
}

template <typename index_t, typename value_t>
__global__ void propogate_spmm_cuda_kernel(
    index_t n_nodes, index_t n_features,
    const index_t *indptr, const index_t *indices,
    const value_t *values, const value_t *features,
    value_t *out_features) {
    for (index_t row = blockIdx.x; row < n_nodes; row += gridDim.x) {
        for (index_t k = threadIdx.x; k < n_features; k += blockDim.x) {
            value_t out = 0.0;
            for (index_t i = indptr[row]; i < indptr[row + 1]; i += 1) {
                index_t col = indices[i];
                out += values[i] * features[col * n_features + k];
            }
            out_features[row * n_features + k] = out;
        }
    }
}

torch::Tensor propogate_spmm_cuda(const torch::Tensor &indptr,
                                  const torch::Tensor &indices,
                                  const torch::Tensor &values,
                                  const torch::Tensor &features) {
    TORCH_CHECK(indptr.dim() == 1);
    TORCH_CHECK(features.dim() == 2);
    int32_t n_nodes = indptr.size(0) - 1;
    int32_t n_features = features.size(1);

    auto new_features = torch::zeros({n_nodes, n_features}, features.options());

    propogate_spmm_cuda_kernel<int32_t, float><<<n_nodes, min(32, n_features)>>>(
        n_nodes, n_features, indptr.data_ptr<int32_t>(), indices.data_ptr<int32_t>(),
        values.data_ptr<float>(), features.data_ptr<float>(), new_features.data_ptr<float>());

    return new_features;
}
